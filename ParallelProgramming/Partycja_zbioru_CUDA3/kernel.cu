#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>

using namespace std;

void scal(int* tab, int p1, int p2, int koniec, int* pom)
{
    int i = p1;
    int j = p2;
    int k = 0;
    while (i != p2 && j != koniec)
    {
        if (tab[i] > tab[j])
        {
            pom[k] = tab[i];
            i++;
            k++;
        }
        else
        {
            pom[k] = tab[j];
            j++;
            k++;
        }
    }
    while (i != p2)
    {
        pom[k] = tab[i];
        i++;
        k++;
    }
    while (j != koniec)
    {
        pom[k] = tab[j];
        j++;
        k++;
    }
    for (int a = 0; a < koniec - p1; a++)
        tab[a + p1] = pom[a];
}

void mergesort(int* tab, int p, int koniec, int* pom)
{
    if (koniec - p < 2)
        return;
    int s = (p + koniec) / 2;
    mergesort(tab, p, s, pom);
    mergesort(tab, s, koniec, pom);
    scal(tab, p, s, koniec, pom);
}

hipError_t addWithCuda(int* c1, int* c2, int* c3, int* c4, int* c5, int* c6, int* c7, int* c8, int* c9, int* size_of_set, const int* a, const int* b, unsigned int size);

__global__ void ParallelGreedyPartition(int* used2, int* used3, int* used4, int* used5, int* used6, int* used7, int* used8, int* used9, int* used10, int* tab_size_of_set, const int* Set, const int* tab_division)
{
    int id = threadIdx.x;
    int set_size = tab_size_of_set[0];
    int upto_division = tab_division[0];
    int division = id + 2;
    //if (id + 2 <= upto_division)
    //    division = id + 2;
    //else
    //    division = 999999;

    int** used = new int* [9];
    for (int i = 0; i < 9; i++) {
        used[i] = new int[set_size];
    }
    used[0] = used2;
    used[1] = used3;
    used[2] = used4;
    used[3] = used5;
    used[4] = used6;
    used[5] = used7;
    used[6] = used8;
    used[7] = used9;
    used[8] = used10;


    if (division <= upto_division)
    {
        int sum = 0;
        for (int i = 0; i < set_size; i++)
            sum = sum + Set[i];

        bool possible_division = false;

        if (sum % division == 0)
        {
            int subsum = sum / division;

            int** tab = new int* [subsum + 1];
            for (int i = 0; i < subsum + 1; i++) {
                tab[i] = new int[set_size + 1];
            }

            for (int d = 0; d < division - 1; d++)
            {
                for (int i = 0; i < set_size + 1; i++)
                    tab[0][i] = 1;
                for (int i = 1; i < subsum + 1; i++)
                {
                    for (int j = 0; j < set_size + 1; j++)
                        tab[i][j] = 0;
                }
                for (int i = 1; i < subsum + 1; i++)
                {
                    for (int j = 1; j < set_size + 1; j++)
                    {
                        tab[i][j] = tab[i][j - 1];
                        if (tab[i][j] == 0 && i >= Set[j - 1] && used[id][j - 1] == 0)
                            tab[i][j] = tab[i - Set[j - 1]][j - 1];
                    }
                }
                int k = subsum;
                possible_division = false;
                while (k > 0)
                {
                    for (int j = 1; j < set_size + 1; j++)
                    {
                        if (tab[k][j] == 1)
                        {
                            used[id][j - 1] = d + 1;
                            possible_division = true;
                            k = k - Set[j - 1];
                            break;
                        }
                    }
                    if (possible_division == false)
                        break;
                }
                if (possible_division == false)
                    break;
            }
        }


        if (possible_division == false)
        {
            for (int i = 0; i < set_size; i++)
                used[id][i] = -1;
        }
    }
}

int main()
{
    int const number_of_tests = 1000;//2;//1000;
    int const size_of_set = 100;//10;//100
    int const max_value = 30;//4;//30
    int const seed = 1;//2;//1;
    int const division = 10;
    int Set[size_of_set];
    int pom[size_of_set];
    bool display = false;
    double greedy_partition_time[number_of_tests];

    int tab_size_of_set[size_of_set] = { size_of_set };
    int tab_division[size_of_set] = { division };

    int a[size_of_set] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10 };
    int b[size_of_set] = { 10, 20, 30, 40, 50, 60, 70, 80, 90, 100 };

    int used2[size_of_set] = { 0 };
    int used3[size_of_set] = { 0 };
    int used4[size_of_set] = { 0 };
    int used5[size_of_set] = { 0 };
    int used6[size_of_set] = { 0 };
    int used7[size_of_set] = { 0 };
    int used8[size_of_set] = { 0 };
    int used9[size_of_set] = { 0 };
    int used10[size_of_set] = { 0 };

    int** used = new int* [9];
    for (int i = 0; i < 9; i++) {
        used[i] = new int[size_of_set];
    }
    used[0] = used2;
    used[1] = used3;
    used[2] = used4;
    used[3] = used5;
    used[4] = used6;
    used[5] = used7;
    used[6] = used8;
    used[7] = used9;
    used[8] = used10;

    srand(seed);


    for (int i = 0; i < number_of_tests; i++)
    {
        for (int k = 0; k < size_of_set; k++)
            Set[k] = rand() % max_value + 1;

        if (display == true)
        {
            cout << "Set"<< i + 1 << ": ";
            for (int k = 0; k < size_of_set; k++)
                cout << Set[k] << " ";
            cout << endl;
            cout << endl;
        }

        auto start = std::chrono::high_resolution_clock::now();

        mergesort(Set, 0, size_of_set, pom);
        hipError_t cudaStatus = addWithCuda(used2, used3, used4, used5, used6, used7, used8, used9, used10, tab_size_of_set, Set, tab_division, size_of_set);

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        greedy_partition_time[i] = elapsed.count();


        //if (cudaStatus != hipSuccess) {
        //    fprintf(stderr, "addWithCuda failed!");
        //     return 1;
        //}
        if (i == number_of_tests - 1)
        {
            cudaStatus = hipDeviceReset();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceReset failed!");
                return 1;
            }
        }

        if (display == true)
        {
            cout << "Sorted_Set: ";
            for (int k = 0; k < size_of_set; k++)
                cout << Set[k] << " ";
            cout << endl;
            cout << endl;
            cout << endl;

            for (int k = 0; k < division - 1; k++)
            {
                cout << "Parallel_Greedy_Partition into " << k + 2 << " subsets: " << endl;
                for (int d = 0; d < k + 2; d++)
                {
                    if (used[k][0] == -1)
                    {
                        cout << "Partition impossible" << endl;
                        break;
                    }
                    cout << "Subset " << d + 1 << ": ";
                    for (int j = 0; j < size_of_set; j++)
                    {
                        if (used[k][j] == d)
                            cout << Set[j] << " ";
                    }
                    cout << endl;
                }
                cout << endl;
            }
            cout << endl;
            cout << endl;
        }
    }

    double greedy_partition_time_sum = 0;
    for (int i = 0; i < number_of_tests; i++)
        greedy_partition_time_sum += greedy_partition_time[i];
    cout << "time_sum: " << greedy_partition_time_sum << endl;

    return 0;
}

hipError_t addWithCuda(int* c1, int* c2, int* c3, int* c4, int* c5, int* c6, int* c7, int* c8, int* c9, int* size_of_set, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c1 = 0;
    int* dev_c2 = 0;
    int* dev_c3 = 0;
    int* dev_c4 = 0;
    int* dev_c5 = 0;
    int* dev_c6 = 0;
    int* dev_c7 = 0;
    int* dev_c8 = 0;
    int* dev_c9 = 0;
    int* dev_size_of_set = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c1, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c2, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c3, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c4, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c5, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c6, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c7, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c8, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c9, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_size_of_set, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_size_of_set, size_of_set, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    ParallelGreedyPartition << <1, size >> > (dev_c1, dev_c2, dev_c3, dev_c4, dev_c5, dev_c6, dev_c7, dev_c8, dev_c9, dev_size_of_set, dev_a, dev_b);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(c1, dev_c1, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c2, dev_c2, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c3, dev_c3, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c4, dev_c4, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c5, dev_c5, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c6, dev_c6, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c7, dev_c7, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c8, dev_c8, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    cudaStatus = hipMemcpy(c9, dev_c9, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c1);
    hipFree(dev_c2);
    hipFree(dev_c3);
    hipFree(dev_c4);
    hipFree(dev_c5);
    hipFree(dev_c6);
    hipFree(dev_c7);
    hipFree(dev_c8);
    hipFree(dev_c9); 
    hipFree(dev_size_of_set);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

